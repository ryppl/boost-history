#include "hip/hip_runtime.h"
/*
 * phase_osc_chain.cu
 *
 *  Created on: Apr 1, 2011
 *      Author: mario
 */

/*
 * This example shows how to use odeint on CUDA devices with thrust.
 * Note that we require at least Version 3.2 of the nVidia CUDA SDK
 * and the thrust library should be installed in the CUDA include
 * folder.
 *
 * As example we use a chain of phase oscillators with nearest neighbour
 * coupling, as described in:
 *
 * Avis H. Cohen, Philip J. Holmes and Richard H. Rand:
 * JOURNAL OF MATHEMATICAL BIOLOGY Volume 13, Number 3, 345-369,
 *
 */

#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>

#include <thrust/device_vector.h>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

using namespace std;

using namespace boost::numeric::odeint;

//change this to float if your device does not support double computation
typedef double value_type;

//change this to host_vector< ... > of you want to run on CPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
//typedef thrust::host_vector< value_type > state_type;
//typedef thrust::host_vector< size_t > index_vector_type;


/*
 * This implements the rhs of the dynamical equation:
 * \phi'_0 = \omega_0 + sin( \phi_1 - \phi_0 )
 * \phi'_i  = \omega_i + sin( \phi_i+1 - \phi_i ) + sin( \phi_i - \phi_i-1 )
 * \phi'_N-1 = \omega_N-1 + sin( \phi_N-1 - \phi_N-2 )
 */
class phase_oscillators
{

public:

    struct sys_functor
    {
        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )
        {
            const value_type phi = thrust::get<0>(t);
            const value_type phi_left = thrust::get<1>(t);
            const value_type phi_right = thrust::get<2>(t);
            const value_type omega = thrust::get<3>(t);
            // the dynamical equation
            thrust::get<4>(t) = omega + sin( phi_right - phi ) + sin( phi - phi_left );
        }
    };

    phase_oscillators( state_type &omega )
        : m_omega( omega ) , m_N( omega.size() ) , m_prev( m_N ) , m_next( m_N )
    {
        // build indices pointing to left and right neighbours
        thrust::counting_iterator<size_t> c( 0 );
        thrust::copy( c , c+m_N-1 , m_prev.begin()+1 );
        m_prev[0] = 0; // m_prev = { 0 , 0 , 1 , 2 , 3 , ... , N-1 }

        thrust::copy( c+1 , c+m_N , m_next.begin() );
        m_next[m_N-1] = m_N-1; // m_next = { 1 , 2 , 3 , ... , N-1 , N-1 }

        /*thrust::copy( m_prev.begin() , m_prev.end() ,
                    std::ostream_iterator< size_t >(std::cout, " ") );
        std::cout << std::endl;*/
    }



    void operator() ( const state_type &x , state_type &dxdt , const value_type dt )
    {
        thrust::for_each(
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.begin() ,
                                thrust::make_permutation_iterator( x.begin() , m_prev.begin() ) ,
                                thrust::make_permutation_iterator( x.begin() , m_next.begin() ) ,
                                m_omega.begin() ,
                                dxdt.begin()
                                ) ),
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.end() ,
                                thrust::make_permutation_iterator( x.begin() , m_prev.end() ) ,
                                thrust::make_permutation_iterator( x.begin() , m_next.end() ) ,
                                m_omega.end() ,
                                dxdt.end()) ) ,
                sys_functor()
                );
    }

private:
    const state_type &m_omega;
    const size_t m_N;
    index_vector_type m_prev;
    index_vector_type m_next;
};


const size_t N = 16;
const value_type epsilon = 6.0/(N*N); // should be < 8/N^2 to see phase locking

int main( int arc , char* argv[] )
{
    srand( time(NULL) );
    // create initial conditions on host:
    vector< value_type > x_host( N );
    //create omegas on host
    vector< value_type > omega_host( N );
    for( size_t i=0 ; i<N ; ++i )
    {
        x_host[i] = 2.0*3.14159265*(double)(rand())/RAND_MAX;
        omega_host[i] = (N-i)*epsilon; // decreasing frequencies
    }

    //copy to device
    state_type x = x_host;
    state_type omega = omega_host;

    //create error stepper
    explicit_rk4< state_type , value_type , state_type , value_type ,
                  thrust_algebra , thrust_operations , adjust_size_initially_tag  > stepper;

    phase_oscillators sys( omega );

    value_type t = 0.0;
    const value_type dt = 0.1;
    while( t < 10.0 )
    {
        stepper.do_step( sys , x , t , dt );
        t += dt;
    }

    /**ToDo: use integrate functions, maybe with algebra_dispatcher */

    //perform integration using standard Runge-Kutta-Cash-Carp Stepper and error bounds ~ 1E-6
    //integrate_const( phase_oscillators(omega) , x , 0.0 , 100.0 , 0.1 );

    thrust::copy( x.begin() , x.end() ,
            std::ostream_iterator< value_type >(std::cout, " ") );
    std::cout << std::endl;
}
